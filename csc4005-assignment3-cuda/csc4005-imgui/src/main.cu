#include "hip/hip_runtime.h"
#include <graphic/graphic.hpp>
#include <chrono>
#include <imgui_impl_sdl.h>
#include <cstring>
#include <nbody/bodycu_thread.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

template <typename ...Args>
void UNUSED(Args&&... args [[maybe_unused]]) {}
static constexpr size_t SHOW_THRESHOLD = 500000000ULL;

__host__ int get_length(int num, int thd, int idx) {
    return ((num - idx) % thd > 0) + (num - idx) / thd;
}

__global__ void mykernel(
    double* d_arr_x,  
    double* d_arr_y, 
    double* d_arr_vx, 
    double* d_arr_vy,
    double* d_arr_ax,
    double* d_arr_ay,
    double* d_arr_m,
    int bodies,
    double elapse,
    double gravity,
    double position_range,
    double radius,
    int* acc_size,
    int* start_pos) {

    pool_update_for_tick(
    d_arr_x,  
    d_arr_y, 
    d_arr_vx, 
    d_arr_vy,
    d_arr_ax,
    d_arr_ay,
    d_arr_m,
    bodies, 
    elapse,
    gravity,
    position_range,
    radius,
    acc_size,
    start_pos);
}


int main(int argc, char **argv) {
    UNUSED(argc, argv);
    static float gravity = 100;
    static float space = 800;
    static float radius = 5;
    static int bodies = 200;
    static float elapse = 0.01;
    static ImVec4 color = ImVec4(1.0f, 1.0f, 0.4f, 1.0f);
    static float max_mass = 50;
    int num_thd;
    int n_bodies = 0;
    int duration = 0;
    // int mode;

    if (argc < 3) {
        std::cerr << "wrong arguments" << std::endl;
        std::cerr << "usage: " << argv[0] << " <integer-bodies> <integer-thread>" << std::endl;
        exit(1);
    }

    sscanf(argv[1], "%d", &bodies);
    printf("bodies = %d\n", bodies);

    sscanf(argv[2], "%d", &num_thd);
    printf("threads = %d\n", num_thd);

    static float current_space = space;
    static float current_max_mass = max_mass;
    static int current_bodies = bodies;

    BodyPool pool(static_cast<size_t>(bodies), space, max_mass);
    graphic::GraphicContext context{"Assignment 2"};
    context.run([&](graphic::GraphicContext *context [[maybe_unused]], SDL_Window *) {
        auto io = ImGui::GetIO();
        ImGui::SetNextWindowPos(ImVec2(0.0f, 0.0f));
        ImGui::SetNextWindowSize(io.DisplaySize);
        ImGui::Begin("Assignment 2", nullptr,
                     ImGuiWindowFlags_NoMove
                     | ImGuiWindowFlags_NoCollapse
                     | ImGuiWindowFlags_NoTitleBar
                     | ImGuiWindowFlags_NoResize);
        ImDrawList *draw_list = ImGui::GetWindowDrawList();
        ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate,
                    ImGui::GetIO().Framerate);
        ImGui::DragFloat("Space", &current_space, 10, 200, 1600, "%f");
        ImGui::DragFloat("Gravity", &gravity, 0.5, 0, 1000, "%f");
        ImGui::DragFloat("Radius", &radius, 0.5, 2, 20, "%f");
        ImGui::DragInt("Bodies", &current_bodies, 1, 2, 100, "%d");
        ImGui::DragFloat("Elapse", &elapse, 0.001, 0.001, 10, "%f");
        ImGui::DragFloat("Max Mass", &current_max_mass, 0.5, 5, 100, "%f");
        ImGui::ColorEdit4("Color", &color.x);
        if (current_space != space || current_bodies != bodies || current_max_mass != max_mass) {
            space = current_space;
            bodies = current_bodies;
            max_mass = current_max_mass;
            pool = BodyPool{static_cast<size_t>(bodies), space, max_mass};
        }
        {   
            const ImVec2 p = ImGui::GetCursorScreenPos();

            auto begin = std::chrono::high_resolution_clock::now();
            if (num_thd > bodies) {
                num_thd = bodies;
            }

            int *acc_size = (int *)malloc(sizeof(int) * num_thd); // the row_size of each rank
            int *start_pos = (int *)malloc(sizeof(int) * num_thd); // the start position of each rank
            double *arr_x = (double *)malloc(sizeof(double) * bodies);
            double *arr_y = (double *)malloc(sizeof(double) * bodies);
            double *arr_vx = (double *)malloc(sizeof(double) * bodies);
            double *arr_vy = (double *)malloc(sizeof(double) * bodies);
            double *arr_ax = (double *)malloc(sizeof(double) * bodies);
            double *arr_ay = (double *)malloc(sizeof(double) * bodies);
            double *arr_m = (double *)malloc(sizeof(double) * bodies);

            double *d_arr_x, *d_arr_y, *d_arr_vx, *d_arr_vy, *d_arr_ax, *d_arr_ay, *d_arr_m;
            int *d_acc_size, *d_start_pos;

            hipMalloc((void **)&d_arr_x, sizeof(double) * bodies);
            hipMalloc((void **)&d_arr_y, sizeof(double) * bodies);
            hipMalloc((void **)&d_arr_vx, sizeof(double) * bodies);
            hipMalloc((void **)&d_arr_vy, sizeof(double) * bodies);
            hipMalloc((void **)&d_arr_ax, sizeof(double) * bodies);
            hipMalloc((void **)&d_arr_ay, sizeof(double) * bodies);
            hipMalloc((void **)&d_arr_m, sizeof(double) * bodies);
            hipMalloc((void **)&d_acc_size, sizeof(int) * num_thd);
            hipMalloc((void **)&d_start_pos, sizeof(int) * num_thd);

            int acc_pos = 0;
            for (int i = 0; i < num_thd; i++) {
                acc_size[i] = get_length(bodies, num_thd, i);
                start_pos[i] = acc_pos;
                acc_pos = acc_pos + acc_size[i];
            }

            for (int i = 0; i < bodies; i++) {
                pool.ax[i] = 0;
                pool.ay[i] = 0;
                arr_x[i] = pool.x[i];
                arr_y[i] = pool.y[i];
                arr_vx[i] = pool.vx[i];
                arr_vy[i] = pool.vy[i];
                arr_ax[i] = pool.ax[i];
                arr_ay[i] = pool.ay[i];
                arr_m[i] = pool.m[i];
            }

            hipMemcpy(d_arr_x, arr_x, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_arr_y, arr_y, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_arr_vx, arr_vx, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_arr_vy, arr_vy, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_arr_ax, arr_ax, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_arr_ay, arr_ay, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_arr_m, arr_m, sizeof(double) * bodies, hipMemcpyHostToDevice);
            hipMemcpy(d_acc_size, acc_size, sizeof(int) * num_thd, hipMemcpyHostToDevice);
            hipMemcpy(d_start_pos, start_pos, sizeof(int) * num_thd, hipMemcpyHostToDevice);


            mykernel<<<1,num_thd>>>(d_arr_x, d_arr_y, d_arr_vx, d_arr_vy, d_arr_ax, d_arr_ay, d_arr_m,
                bodies, elapse, gravity, space, radius, d_acc_size, d_start_pos);
            

            hipMemcpy(arr_x, d_arr_x, sizeof(double) * bodies, hipMemcpyDeviceToHost);
            hipMemcpy(arr_y, d_arr_y, sizeof(double) * bodies, hipMemcpyDeviceToHost);
            hipMemcpy(arr_vx, d_arr_vx, sizeof(double) * bodies, hipMemcpyDeviceToHost);
            hipMemcpy(arr_vy, d_arr_vy, sizeof(double) * bodies, hipMemcpyDeviceToHost);
            hipMemcpy(arr_ax, d_arr_ax, sizeof(double) * bodies, hipMemcpyDeviceToHost);
            hipMemcpy(arr_ay, d_arr_ay, sizeof(double) * bodies, hipMemcpyDeviceToHost);
            hipMemcpy(arr_m, d_arr_m, sizeof(double) * bodies, hipMemcpyDeviceToHost);


            for (int i = 0; i < bodies; i++) {
                pool.x[i] = arr_x[i];
                pool.y[i] = arr_y[i];
                pool.vx[i] = arr_vx[i];
                pool.vy[i] = arr_vy[i];
                pool.ax[i] = arr_ax[i];
                pool.ay[i] = arr_ay[i];
                pool.m[i] = arr_m[i];
            }
            auto end = std::chrono::high_resolution_clock::now();
            duration += (end - begin).count();
            n_bodies += bodies;

            if (duration > SHOW_THRESHOLD) {
                std::cout << n_bodies << " bodies in last " << duration << " nanoseconds\n";
                auto speed = static_cast<double>(n_bodies) / static_cast<double>(duration) * 1e9;
                std::cout << "speed: " << speed << " bodies per second" << std::endl;
                n_bodies = 0;
                duration = 0;
            }

            for (int i = 0; i < bodies; ++i) {
                auto x = p.x + static_cast<float>(arr_x[i]);
                auto y = p.y + static_cast<float>(arr_y[i]);            
                draw_list->AddCircleFilled(ImVec2(x, y), radius, ImColor{color});
            }

            // Cleanup
            free(arr_x); free(arr_y); free(arr_vx); free(arr_vy); free(arr_ax); free(arr_ay); free(arr_m); 
            hipFree(d_arr_x); hipFree(d_arr_y); hipFree(d_arr_vx); hipFree(d_arr_vy);hipFree(d_arr_ax);hipFree(d_arr_ay);hipFree(d_arr_m);

        }
        ImGui::End();

        // pool.print(i);

        // i++;

        // if (i == 60) {
        //     exit(0);
        // }

    });
}
